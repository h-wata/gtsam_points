// SPDX-License-Identifier: MIT
// Copyright (c) 2021  Kenji Koide (k.koide@aist.go.jp)

#include <gtsam_points/cuda/check_error.cuh>

namespace gtsam_points {

void CUDACheckError::operator<<(hipError_t error) const {
  if (error == hipSuccess) return;
  std::cerr << "warning: " << hipGetErrorName(error) << "\n"
            << "         : " << hipGetErrorString(error) << "\n";
  if (file_) {
    std::cerr << "  (from " << file_ << ":" << line_ << ")\n";
  }
}

CUDACheckError check_error;

}  // namespace gtsam_points
